
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

__global__ void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1<<20; 
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    
    hipFree(x);
    hipFree(y);

    auto t1 = std::chrono::high_resolution_clock::now();
    add<<<1,256>>>(N, x, y); 

    hipDeviceSynchronize();

    auto t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl; 
    std::cout << duration;
    delete [] x;
    delete [] y;
    return 0;
}

