
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void mykernal(int n, float *x, float *y){
	// if(blockIdx.x == 0)
	//	printf("-Block %d\n", blockIdx.x);
	//printf("-    Block: %d  ||  Thread: %d\n", blockIdx.x, threadIdx.x);
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		y[i] = x[i] + y[i];
}

int main(void)
{
	int N = 1<<20;

	float *x, *y;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	mykernal<<<numBlocks, blockSize>>>(N, x, y);

	// std::cout<<"Hello World!"<<std::endl;
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	std::cout << "Max error: " << maxError << std::endl;

	// Free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
